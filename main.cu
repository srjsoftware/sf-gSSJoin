#include "hip/hip_runtime.h"
/*********************************************************************
11
12	 Copyright (C) 2016 by Sidney Ribeiro Junior
13
14	 This program is free software; you can redistribute it and/or modify
15	 it under the terms of the GNU General Public License as published by
16	 the Free Software Foundation; either version 2 of the License, or
17	 (at your option) any later version.
18
19	 This program is distributed in the hope that it will be useful,
20	 but WITHOUT ANY WARRANTY; without even the implied warranty of
21	 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
22	 GNU General Public License for more details.
23
24	 You should have received a copy of the GNU General Public License
25	 along with this program; if not, write to the Free Software
26	 Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
27
28	 ********************************************************************/

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <vector>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include <map>

#include "structs.cuh"
#include "utils.cuh"
#include "inverted_index.cuh"
#include "simjoin.cuh"
#include "tests.cu"


#define OUTPUT 1
#define NUM_STREAMS 1

using namespace std;


FileStats readInputFile(string &file, vector<Entry> &entries);
void allocVariables(DeviceVariables *dev_vars, Pair **similar_pairs, int num_terms, int block_size, int entries_size, int num_sets);
void freeVariables(DeviceVariables *dev_vars, Pair **similar_pairs);
void write_output(Pair *similar_pairs, int totalSimilars, stringstream &outputfile);

/**
 * Receives as parameters the training file name and the test file name
 */
int main(int argc, char **argv) {

	if (argc != 6) {
		cerr << "Wrong parameters. Correct usage: <executable> <input_file> <threshold> <output_file> <number_of_gpus> <size of blocks>" << endl;
		exit(1);
	}

	vector<Entry> entries;
	float threshold = atof(argv[2]);
	int gpuNum;

	hipGetDeviceCount(&gpuNum);
	if (gpuNum > atoi(argv[4]) && atoi(argv[4]) > 0)
		gpuNum = atoi(argv[4]);
	omp_set_num_threads(gpuNum);

	string inputFileName(argv[1]);
	printf("Reading file %s...\n", inputFileName.c_str());
	FileStats stats = readInputFile(inputFileName, entries);

	ofstream ofsf(argv[3], ofstream::trunc);
	ofsf.close();
	ofstream ofsfileoutput(argv[3], ofstream::out | ofstream::app);
	vector<stringstream*> outputString; // Each thread has an output string.
	for (int i = 0; i < gpuNum; i++)
		outputString.push_back(new stringstream);

	// calculating maximum size of data structures
	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);
	long sizeEntries = (stats.start[stats.num_sets - 1] + stats.sizes[stats.num_sets - 1]) * sizeof(Entry);
	long sizeVectorsN = stats.num_sets*sizeof(int);
	long freeMem = free_mem - 3*sizeEntries - sizeVectorsN*4;

	int block_size = atoi(argv[5]);
	block_size = block_size < 1? freeMem / (stats.num_sets*(sizeof(float) + sizeof(Pair))): block_size;
	block_size = block_size > stats.num_sets? stats.num_sets: block_size;
	int block_num = ceil((float) stats.num_sets / block_size);

	double start = gettime();

	#pragma omp parallel num_threads(gpuNum)
	{
		int gpuid = omp_get_thread_num();
		hipSetDevice(gpuid);
		InvertedIndex index;
		DeviceVariables dev_vars;
		Pair *similar_pairs;

		allocVariables(&dev_vars, &similar_pairs, stats.num_terms, block_size, entries.size(), stats.num_sets);
		gpuAssert(hipMemcpy(dev_vars.d_starts, &stats.start[0], stats.num_sets * sizeof(int), hipMemcpyHostToDevice));
		gpuAssert(hipMemcpy(dev_vars.d_sizes, &stats.sizes[0], stats.num_sets * sizeof(int), hipMemcpyHostToDevice));

		for (int i = gpuid; i < block_num; i+= gpuNum) {
			int entries_block_start = i*block_size;
			int entries_offset = stats.start[entries_block_start];
			int last_set = entries_block_start + block_size >= stats.num_sets? stats.num_sets - 1: entries_block_start + block_size - 1;
			int entries_block_size = last_set - entries_block_start + 1;
			int entries_size = stats.start[last_set] + stats.sizes[last_set] - entries_offset;
			//printf("=========Indexed Block %d=========\nset_offset = %d\nentrie_offset: %d\nlast_set: %d\nentries_size: %d\n", i, entries_block_start, entries_offset, last_set, entries_size);

			// build the inverted index for block i of size block_size
			index = make_inverted_index(stats.num_sets, stats.num_terms, entries_size, entries_offset, entries, &dev_vars);
			//print_sets(entries, stats.sizes, stats.start); //print_invertedIndex(index);

			for (int j = 0; j <= i; j++) { // calculate similarity between indexed sets and probe sets
				int probe_block_start = j*block_size;
				int last_probe = probe_block_start + block_size > stats.num_sets? stats.num_sets - 1: probe_block_start + block_size - 1;
				int probe_block_size = last_probe - probe_block_start + 1;
				int probes_offset = stats.start[probe_block_start];

				// size filtering
				if (stats.sizes[last_probe] < threshold * stats.sizes[entries_block_start])
					continue;

				if (j < i) {
					int probes_size = stats.start[last_probe] + stats.sizes[last_probe] - probes_offset;
					gpuAssert(hipMemcpy(dev_vars.d_probes, &entries[probes_offset], probes_size * sizeof(Entry), hipMemcpyHostToDevice));
				}
				//printf("=========Probe Block %d=========\nprobe_block_start = %d\nprobe_offset: %d\nlast_probe: %d\nprobe_block_size: %d\n===============================\n", j, probe_block_start, probes_offset,last_probe, probe_block_size);

				int totalSimilars = findSimilars(index, threshold, &dev_vars, similar_pairs, probe_block_start,
						probe_block_size, probes_offset, entries_block_size, entries_block_start, i, j);

				//print_intersection(dev_vars.d_intersection, block_size, i, j);
				//print_result(similar_pairs, totalSimilars);
				write_output(similar_pairs, totalSimilars, *outputString[gpuid]);
			}

		}

		freeVariables(&dev_vars, &similar_pairs);
	}

	double end = gettime();

	printf("Time to process similarity join between %d sets: %lf seconds\n", stats.num_sets, end - start);

	for (int i = 0; i < gpuNum; i++)
		ofsfileoutput << outputString[i]->str();
	ofsfileoutput.close();

	return 0;
}

FileStats readInputFile(string &filename, vector<Entry> &entries) {
	ifstream input(filename.c_str());
	string line;

	FileStats stats;
	int accumulatedsize = 0;
	int doc_id = 0;

	while (!input.eof()) {
		getline(input, line);
		if (line == "") continue;

		vector<string> tokens = split(line, ' ');
		//biggestQuerySize = max((int)tokens.size() / 2, biggestQuerySize);

		int size = (tokens.size() - 2)/2;
		stats.sizes.push_back(size);
		stats.start.push_back(accumulatedsize);
		accumulatedsize += size;

		for (int i = 2, size = tokens.size(); i + 1 < size; i += 2) {
			int term_id = atoi(tokens[i].c_str());
			int term_count = atoi(tokens[i + 1].c_str());
			stats.num_terms = max(stats.num_terms, term_id + 1);
			entries.push_back(Entry(doc_id, term_id, term_count));
		}
		doc_id++;
	}

	stats.num_sets = stats.start.size();

	input.close();

	return stats;
}

void allocVariables(DeviceVariables *dev_vars, Pair **similar_pairs, int num_terms, int block_size, int entries_size, int num_sets) {
	// TODO alocar o tamanho certo para entries, probes e o índice invertido

	// Inverted index's variables
	gpuAssert(hipMalloc(&dev_vars->d_inverted_index, entries_size * sizeof(Entry)));
	gpuAssert(hipMalloc(&dev_vars->d_entries, entries_size * sizeof(Entry)));
	gpuAssert(hipMalloc(&dev_vars->d_index, num_terms * sizeof(int)));
	gpuAssert(hipMalloc(&dev_vars->d_count, num_terms * sizeof(int)));

	// Variables used to perform the similarity join
	gpuAssert(hipMalloc(&dev_vars->d_probes, entries_size * sizeof(Entry)));
	gpuAssert(hipMalloc(&dev_vars->d_intersection, (1 + block_size * block_size) * sizeof(int)));
	gpuAssert(hipMalloc(&dev_vars->d_pairs, block_size *block_size * sizeof(Pair)));
	gpuAssert(hipMalloc(&dev_vars->d_sizes, num_sets * sizeof(int)));
	gpuAssert(hipMalloc(&dev_vars->d_starts, num_sets * sizeof(int)));

	*similar_pairs = (Pair *)malloc(sizeof(Pair)*block_size*block_size);
}

void freeVariables(DeviceVariables *dev_vars, Pair **similar_pairs) {
	hipFree(&dev_vars->d_inverted_index);
	hipFree(&dev_vars->d_entries);
	hipFree(&dev_vars->d_index);
	hipFree(&dev_vars->d_count);

	hipFree(&dev_vars->d_probes);
	hipFree(&dev_vars->d_intersection);
	hipFree(&dev_vars->d_pairs);
	hipFree(&dev_vars->d_sizes);
	hipFree(&dev_vars->d_starts);

	free(*similar_pairs);
}

void write_output(Pair *similar_pairs, int totalSimilars, stringstream &outputfile) {
	for (int i = 0; i < totalSimilars; i++) {
		outputfile << "(" << similar_pairs[i].set_x << ", " << similar_pairs[i].set_y << "): " << similar_pairs[i].similarity << endl;
	}
}
